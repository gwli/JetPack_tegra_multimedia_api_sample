#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include <EGL/egl.h>
#include <EGL/eglext.h>
#include <GLES2/gl2.h>
#include <GLES2/gl2ext.h>

#include "cudaEGL.h"
#include "NvAnalysis.h"

#include "v4l2_backend_test.h"

#define BOX_W 32
#define BOX_H 32

__constant__ int sample_count = 4;
__constant__ int samples[] =
{
        0,0,
        1,0,
        0,1,
        1,1,
};

__global__ void
prepareForTensorRtKernelRGB(hipDeviceptr_t pDevPtr,
        int src_width, int src_height, int src_pitch,
        int dst_width, int dst_height,
        void* cuda_buf)
{
    float *pdata = (float *)cuda_buf;
    char *psrcdata = (char *)pDevPtr;
    int dst_row = blockIdx.y * blockDim.y + threadIdx.y;
    int dst_col = blockIdx.x * blockDim.x + threadIdx.x;

    if (dst_col < dst_width && dst_row < dst_height)
    {
        int src_row = dst_row*2;
        int src_col = dst_col*2;

        for (int channel = 0; channel < 3; channel++)
        {
            int accum = 0;
            for (int s = 0; s < sample_count; ++s)
            {
                int offX=samples[s*2+0];
                int offY=samples[s*2+1];
                accum += (int)*(psrcdata + (src_row+offX) * src_pitch + (src_col+offY) * 4 + (3 - 1 - channel));
            }

            pdata[dst_width * dst_height * channel + dst_row * dst_width + dst_col] = (1.0f/sample_count)*accum;
        }
    }
}

__global__ void
prepareForTensorRtKernelBGR(hipDeviceptr_t pDevPtr,
        int src_width, int src_height, int src_pitch,
        int dst_width, int dst_height,
        void* cuda_buf)
{
    float *pdata = (float *)cuda_buf;
    char *psrcdata = (char *)pDevPtr;
    int dst_row = blockIdx.y * blockDim.y + threadIdx.y;
    int dst_col = blockIdx.x * blockDim.x + threadIdx.x;

    // BGR offset for 3 classes
    int offsets[] = {124, 117, 104};

    if (dst_col < dst_width && dst_row < dst_height)
    {
        int src_row = dst_row*2;
        int src_col = dst_col*2;

        // For V4L2_PIX_FMT_ABGR32 --> BGRA-8-8-8-8
        for (int k = 0; k < 3; k++)
        {
            int accum = 0;
            for (int s = 0; s < sample_count; ++s)
            {
                int offX=samples[s*2+0];
                int offY=samples[s*2+1];
                accum += (int)*(psrcdata + (src_row+offX) * src_pitch + (src_col+offY) * 4 + k) - offsets[k];
            }

            pdata[dst_width * dst_height * k + dst_row * dst_width + dst_col] = (1.0f/sample_count)*accum;

        }
    }
}

int prepareForTensorRt(hipDeviceptr_t pDevPtr,
                      int src_width,
                      int src_height,
                      int src_pitch,
                      int dst_width,
                      int dst_height,
                      COLOR_FORMAT color_format,
                      void* cuda_buf)
{
    dim3 threadsPerBlock(32, 32);
    dim3 blocks(dst_width/threadsPerBlock.x, dst_height/threadsPerBlock.y);


    if (color_format == COLOR_FORMAT_RGB)
    {
        prepareForTensorRtKernelRGB<<<blocks, threadsPerBlock>>>(pDevPtr,
                src_width, src_height, src_pitch, dst_width, dst_height,
                cuda_buf);
    }
    else if (color_format == COLOR_FORMAT_BGR)
    {
        prepareForTensorRtKernelBGR<<<blocks, threadsPerBlock>>>(pDevPtr,
                src_width, src_height, src_pitch, dst_width, dst_height,
                cuda_buf);
    }

    return 0;
}

/**
  * Performs map egl image into cuda memory.
  *
  * @param pEGLImage: EGL image
  * @param width: Image width
  * @param height: Image height
  * @param color_format: The input color format
  * @param cuda_buf: destnation cuda address
  */
bool prepareEGLImage2FloatForTensorRt(void* pEGLImage, int width, int height, COLOR_FORMAT color_format, void* cuda_buf)
{
    hipError_t status;
    CUeglFrame eglFrame;
    hipGraphicsResource_t pResource = NULL;
    EGLImageKHR *pImage = (EGLImageKHR *)pEGLImage;

    hipFree(0);
    status = cuGraphicsEGLRegisterImage(&pResource, *pImage,
                CU_GRAPHICS_MAP_RESOURCE_FLAGS_NONE);
    if (status != hipSuccess)
    {
        printf("cuGraphicsEGLRegisterImage failed: %d, cuda process stop\n",
                        status);
        return false;
    }

    status = cuGraphicsResourceGetMappedEglFrame(&eglFrame, pResource, 0, 0);
    if (status != hipSuccess)
    {
        printf("hipGraphicsSubResourceGetMappedArray failed\n");
        return false;
    }

    status = hipCtxSynchronize();
    if (status != hipSuccess)
    {
        printf("hipCtxSynchronize failed\n");
        return false;
    }

    if (eglFrame.frameType == CU_EGL_FRAME_TYPE_PITCH)
    {
        hipDeviceptr_t eglSrcPtr = (hipDeviceptr_t) eglFrame.frame.pPitch[0];

        // Using GPU to convert int buffer into float buffer.
        prepareForTensorRt(
                          eglSrcPtr,
                          eglFrame.width,
                          eglFrame.height,
                          eglFrame.pitch,
                          width,
                          height,
                          color_format,
                          cuda_buf);
    }
    status = hipCtxSynchronize();
    if (status != hipSuccess)
    {
        printf("hipCtxSynchronize failed after prepareForTensorRt\n");
        return false;
    }

    status = hipGraphicsUnregisterResource(pResource);
    if (status != hipSuccess)
    {
        printf("cuGraphicsEGLUnRegisterResource failed: %d\n", status);
        return false;
    }

    return true;
}
